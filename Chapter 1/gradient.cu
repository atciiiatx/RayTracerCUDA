
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>

const int ImageWidth = 1024;
const int ImageHeight = 512;

struct PixelColor
{
    float r;
    float g;
    float b;
};

// Kernel function to add the elements of two arrays
__global__
void CalculatePixelColors(int width, int height, PixelColor* pixels)
{
    int k = 0;
    for (int j = height - 1; j >= 0; --j)
    {
        for (int i = 0; i < width; ++i)
        {
            pixels[k].r = static_cast<float>(i) / static_cast<float>(width);
            pixels[k].g = static_cast<float>(j) / static_cast<float>(height);
            pixels[k].b = 0.2;
            k++;
        }
    }
}

void SaveImage(std::string fileName, int width, int height, PixelColor* pixels)
{
  std::ofstream imageFile;
  imageFile.open(fileName.c_str());
  imageFile << "P3" << std::endl  << width << " " << height << std::endl << 255 << std::endl;
  int k = 0;
  for (int j = 0; j < height; ++j)
  {
      for (int i = 0; i < width; ++i)
      {
          int ir = static_cast<int>(255.99 * pixels[k].r);
          int ig = static_cast<int>(255.99 * pixels[k].g);
          int ib = static_cast<int>(255.99 * pixels[k].b);
          imageFile << ir << " " << ig << " " << ib << std::endl;
          k++;
      }
  }
  imageFile.close();
}

int main(int argc, char** argv)
{
  // Get image file name
  std::string fileName = "gradient.ppm";
  if (argc > 1)
  {
      fileName = argv[1];
  }

  // Allocate Unified Memory – accessible from CPU or GPU
  int numPixels = ImageWidth*ImageHeight;
  PixelColor *pixels;
  hipMallocManaged(&pixels, numPixels*sizeof(PixelColor));

  // Run kernel on 1M elements on the GPU
  CalculatePixelColors<<<1, 1>>>(ImageWidth, ImageHeight, pixels);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  SaveImage(fileName, ImageWidth, ImageHeight, pixels);

  // Free memory
  hipFree(pixels);
  
  return 0;
}